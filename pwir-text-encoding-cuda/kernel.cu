#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <windows.h>
#include <chrono>
#include <thread>
#include <vector>
#include <omp.h>
#include <stdio.h>

#define MAX_TAB_SIZE 1000 
using namespace std;

void ShowIntroInformation(HANDLE hConsole);
void SetSortOption(HANDLE hConsole, int& sort);
void SetText(HANDLE hConsole, char text[MAX_TAB_SIZE]);
void SetKeyValue(HANDLE hConsole, int& key);
void SetDelayValue(HANDLE hConsole, int& delay);
void SetDisplayValue(HANDLE hConsole, int& display);
void SetRepeatValue(HANDLE hConsole, int& repeat);
void RunEncodeTextMethod(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunEncodeTextMethodParallel(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunEncodeTextMethodParallelOpenMP(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunEncodeTextMethodParallelCuda(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunDecodeTextMethod(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunDecodeTextMethodParallel(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunDecodeTextMethodParallelOpenMP(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);
void RunDecodeTextMethodParallelCuda(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display);

int main()
{
	setlocale(LC_CTYPE, "Polish");
	char text[MAX_TAB_SIZE], tmptext[MAX_TAB_SIZE];
	int sort, key, delay, display, repeat;
	HANDLE hConsole = GetStdHandle(STD_OUTPUT_HANDLE);
	ShowIntroInformation(hConsole);
	while (true)
	{
		SetSortOption(hConsole, sort);
		cin.ignore(MAX_TAB_SIZE, '\n');
		SetText(hConsole, text);
		SetKeyValue(hConsole, key);
		SetDelayValue(hConsole, delay);
		SetDisplayValue(hConsole, display);
		if (sort == 1)
		{
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunEncodeTextMethod(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunEncodeTextMethodParallel(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunEncodeTextMethodParallelOpenMP(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunEncodeTextMethodParallelCuda(hConsole, tmptext, key, delay, display);
		}
		else if (sort == 2)
		{
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunDecodeTextMethod(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunDecodeTextMethodParallel(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunDecodeTextMethodParallelOpenMP(hConsole, tmptext, key, delay, display);
			memcpy(tmptext, text, MAX_TAB_SIZE);
			RunDecodeTextMethodParallelCuda(hConsole, tmptext, key, delay, display);
		}
		SetRepeatValue(hConsole, repeat);
		if (repeat == 0) break;
	}
}

void ShowIntroInformation(HANDLE hConsole)
{
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n\n  PROGRAMOWANIE WSPÓŁBIEŻNE I ROZPROSZONE 21/22L\n  Rozwiązanie równania różniczkowego zwyczajnego\n  Autor programu: ";
	SetConsoleTextAttribute(hConsole, 15);
	cout << "Kamil Hojka -- 97632\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	cout << "\n";
	SetConsoleTextAttribute(hConsole, 15);
}

void SetSortOption(HANDLE hConsole, int& sort)
{
	SetConsoleTextAttribute(hConsole, 14);
	cout << "\n -> Opcje:";
	cout << "\n --> [1] Szyfrowanie (Encode)";
	cout << "\n --> [2] Deszyfrowanie (Decode)";
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n --> Wybierz spośród dostępnych opcji: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin >> sort;
		if (cin.good() && (sort == 1 || sort == 2)) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wartość musi być liczbą równą jednej z dostępnych opcji\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void SetText(HANDLE hConsole, char text[MAX_TAB_SIZE])
{
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n -> Wprowadź tekst ";
		SetConsoleTextAttribute(hConsole, 4);
		cout << "[Limit znaków: 1000]: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin.getline(text, MAX_TAB_SIZE);
		if (cin.good()) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wystąpił błąd podczas wprowadzenia tekstu, spróbuj jeszcze raz\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void SetKeyValue(HANDLE hConsole, int& key)
{
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n -> Podaj wartość klucza? [-26, ..., 26]: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin >> key;
		if (cin.good() && (key <= 26 && key >= -26)) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wartość klucza musi mieścić się w przedziale [-26, ..., 26]\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void SetDelayValue(HANDLE hConsole, int& delay)
{
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n -> Podaj opóźnienie? [ms]: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin >> delay;
		if (cin.good() && delay >= 0) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wartość opóźnienia musi być liczbą naturalną {0, 1, 2, ...}\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void SetDisplayValue(HANDLE hConsole, int& display)
{
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n -> Czy wyświetlić wynik? [1/0]: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin >> display;
		if (cin.good() && (display == 0 || display == 1)) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wartość musi być liczbą 0 lub 1\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void SetRepeatValue(HANDLE hConsole, int& repeat)
{
	while (true) {
		SetConsoleTextAttribute(hConsole, 14);
		cout << "\n -> Czy powtórzyć program? [1/0]: ";
		SetConsoleTextAttribute(hConsole, 15);
		cin >> repeat;
		if (cin.good() && (repeat == 0 || repeat == 1)) break;
		SetConsoleTextAttribute(hConsole, 4);
		cout << "    ! Wartość musi być liczbą 0 lub 1\n";
		SetConsoleTextAttribute(hConsole, 15);
		cin.clear();
		cin.ignore();
	}
}

void EncodeText(char text[MAX_TAB_SIZE], int key, int delay, int i)
{
	bool isGood = false;
	char a, z;
	this_thread::sleep_for(std::chrono::milliseconds(delay));
	if (text[i] >= 'a' && text[i] <= 'z') {
		a = 'a', z = 'z';
		isGood = true;
	}
	else if (text[i] >= 'A' && text[i] <= 'Z') {
		a = 'A', z = 'Z';
		isGood = true;
	}

	if (isGood) {
		if (key >= 0)
		{
			if (text[i] + key <= z) text[i] += key;
			else text[i] = text[i] + key - 26;
		}
		else {
			if (text[i] + key >= a) text[i] += key;
			else text[i] = text[i] + key + 26;
		}
	}
}

void RunEncodeTextMethod(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Sekwencyjne szyfrowanie tekstu - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
	for (int i = 0; i < strlen(text); i++)
	{
		EncodeText(text, key, delay, i);
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

void RunEncodeTextMethodParallel(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe szyfrowanie tekstu za pomocą thread - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
	vector<thread> threads(strlen(text));
	for (int i = 0; i < threads.size(); i++)
	{
		threads[i] = thread(EncodeText, text, key, delay, i);
	}
	for (auto& thread : threads)
	{
		thread.join();
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

void RunEncodeTextMethodParallelOpenMP(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe szyfrowanie tekstu za pomocą OpenMP - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
#pragma omp parallel for schedule(static, 1)
	for (int i = 0; i < strlen(text); i++)
	{
		EncodeText(text, key, delay, i);
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

void DecodeText(char text[MAX_TAB_SIZE], int key, int delay, int i)
{
	bool isGood = false;
	char a, z;
	this_thread::sleep_for(std::chrono::milliseconds(delay));
	if (text[i] >= 'a' && text[i] <= 'z') {
		a = 'a', z = 'z';
		isGood = true;
	}
	else if (text[i] >= 'A' && text[i] <= 'Z') {
		a = 'A', z = 'Z';
		isGood = true;
	}

	if (isGood) {
		if (key >= 0)
		{
			if (text[i] - key >= a) text[i] -= key;
			else text[i] = text[i] - key + 26;
		}
		else {
			if (text[i] - key <= z) text[i] -= key;
			else text[i] = text[i] - key - 26;
		}
	}
}

void RunDecodeTextMethod(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Sekwencyjne deszyfrowanie tekstu - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
	for (int i = 0; i < strlen(text); i++)
	{
		DecodeText(text, key, delay, i);
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zdeaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

void RunDecodeTextMethodParallel(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe deszyfrowanie tekstu za pomocą thread - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
	vector<thread> threads(strlen(text));
	for (int i = 0; i < threads.size(); i++)
	{
		threads[i] = thread(DecodeText, text, key, delay, i);
	}
	for (auto& thread : threads)
	{
		thread.join();
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zdeaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

void RunDecodeTextMethodParallelOpenMP(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe deszyfrowanie tekstu za pomocą OpenMP - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	auto begin = chrono::high_resolution_clock::now();
#pragma omp parallel for schedule(static, 1)
	for (int i = 0; i < strlen(text); i++)
	{
		DecodeText(text, key, delay, i);
	}
	auto end = chrono::high_resolution_clock::now();
	auto elapsed = chrono::duration_cast<std::chrono::milliseconds>(end - begin).count();
	if (display)
	{
		cout << "\n Zdeszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
}

__global__ void EncodeTextKernel(char* text, int N, int key, int delay)
{
	int i = threadIdx.x;
	if (i < N) {
		bool isGood = false;
		char a, z;
		/*clock_t start = clock();
		clock_t now;
		for (;;) {
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if (cycles >= delay * 1000000) {
				break;
			}
		}*/
		if (text[i] >= 'a' && text[i] <= 'z') {
			a = 'a', z = 'z';
			isGood = true;
		}
		else if (text[i] >= 'A' && text[i] <= 'Z') {
			a = 'A', z = 'Z';
			isGood = true;
		}

		if (isGood) {
			if (key >= 0)
			{
				if (text[i] + key <= z) text[i] += key;
				else text[i] = text[i] + key - 26;
			}
			else {
				if (text[i] + key >= a) text[i] += key;
				else text[i] = text[i] + key + 26;
			}
		}
	}
}

void RunEncodeTextMethodParallelCuda(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe szyfrowanie tekstu za pomocą CUDA - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	char* dev_text;
	int N = strlen(text);
	hipMalloc((void**)&dev_text, N * sizeof(char));
	hipMemcpy(dev_text, text, N * sizeof(char), hipMemcpyHostToDevice);
	dim3 grid_size(1);
	dim3 block_size(N);
	hipEvent_t start, end;
	float elapsed;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);
	this_thread::sleep_for(std::chrono::milliseconds(delay));
	EncodeTextKernel << <grid_size, block_size >> > (dev_text, N, key, delay);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed, start, end);
	hipMemcpy(text, dev_text, N * sizeof(char), hipMemcpyDeviceToHost);
	if (display)
	{
		cout << "\n Zaszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
	hipEventDestroy(start);
	hipEventDestroy(end);
	hipFree(dev_text);
}

__global__ void DecodeTextKernel(char* text, int N, int key, int delay)
{
	int i = threadIdx.x;
	if (i < N) {
		bool isGood = false;
		char a, z;
		/*clock_t start = clock();
		clock_t now;
		for (;;) {
			now = clock();
			clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
			if (cycles >= delay * 1000000) {
				break;
			}
		}*/
		if (text[i] >= 'a' && text[i] <= 'z') {
			a = 'a', z = 'z';
			isGood = true;
		}
		else if (text[i] >= 'A' && text[i] <= 'Z') {
			a = 'A', z = 'Z';
			isGood = true;
		}

		if (isGood) {
			if (key >= 0)
			{
				if (text[i] - key >= a) text[i] -= key;
				else text[i] = text[i] - key + 26;
			}
			else {
				if (text[i] - key <= z) text[i] -= key;
				else text[i] = text[i] - key - 26;
			}
		}
	}
}

void RunDecodeTextMethodParallelCuda(HANDLE hConsole, char text[MAX_TAB_SIZE], int key, int delay, int display)
{
	cout << "\n\n";
	SetConsoleTextAttribute(hConsole, 11);
	for (int i = 0; i < 70; i++) cout << '*';
	SetConsoleTextAttribute(hConsole, 3);
	cout << "\n ---> Równoległe deszyfrowanie tekstu za pomocą CUDA - Szyfr Cezara\n";
	SetConsoleTextAttribute(hConsole, 15);
	char* dev_text;
	int N = strlen(text);
	hipMalloc((void**)&dev_text, N * sizeof(char));
	hipMemcpy(dev_text, text, N * sizeof(char), hipMemcpyHostToDevice);
	dim3 grid_size(1);
	dim3 block_size(N);
	hipEvent_t start, end;
	float elapsed;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);
	DecodeTextKernel << <grid_size, block_size >> > (dev_text, N, key, delay);
	hipEventRecord(end, 0);
	hipEventSynchronize(end);
	hipEventElapsedTime(&elapsed, start, end);
	hipMemcpy(text, dev_text, N * sizeof(char), hipMemcpyDeviceToHost);
	if (display)
	{
		cout << "\n Zdeszyfrowany tekst: ";
		SetConsoleTextAttribute(hConsole, 14);
		cout << text;
		SetConsoleTextAttribute(hConsole, 15);
	}
	cout << "\n\n Zmierzony czas: " << elapsed << " ms\n";
	hipEventDestroy(start);
	hipEventDestroy(end);
	hipFree(dev_text);
}